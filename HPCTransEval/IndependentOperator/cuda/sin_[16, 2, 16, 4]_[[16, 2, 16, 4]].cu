
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}


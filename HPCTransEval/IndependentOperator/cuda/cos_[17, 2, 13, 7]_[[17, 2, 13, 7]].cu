
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(7) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))] = __cosf(data[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))]);
}


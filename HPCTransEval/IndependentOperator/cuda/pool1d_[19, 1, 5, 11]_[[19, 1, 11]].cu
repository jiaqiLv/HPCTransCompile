
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(7) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  if (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) < 114) {
    pool_max[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    if (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) < 114) {
      pool_max[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))], (((1 <= ((((((int)blockIdx.x) + ((int)threadIdx.x)) % 6) * 2) + rv0)) && (((rv0 >> 1) + ((((int)blockIdx.x) + ((int)threadIdx.x)) % 6)) < 6)) ? data[(((((((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) / 6) * 11) + (((((int)blockIdx.x) + ((int)threadIdx.x)) % 6) * 2)) + rv0) - 1)] : -3.402823e+38f));
    }
  }
}


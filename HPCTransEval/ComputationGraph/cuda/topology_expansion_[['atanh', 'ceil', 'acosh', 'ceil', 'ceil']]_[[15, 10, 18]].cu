
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = acoshf(ceilf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}


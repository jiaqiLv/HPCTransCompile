#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

__device__ __forceinline__ float warp_reduce_max(float val) {
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        val = max(val, __shfl_down_sync(FULL_MASK, val, offset));
    }
    return val;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    return val;
}

__global__ void conv_transpose_maxpool_mean_kernel(
    const float* __restrict__ input,
    const float* __restrict__ weight,
    const float* __restrict__ bias,
    float* __restrict__ output,
    float* __restrict__ mean_output,
    int N, int in_channels,
    int H_in, int W_in,
    int out_channels,
    int kernel_h, int kernel_w,
    int stride, int padding,
    int H_out, int W_out,
    int pool_kernel, int pool_stride,
    int H_pool_out, int W_pool_out
) {
    extern __shared__ float shared_mem[];
    float* shared_weight = shared_mem;
    float* shared_reduce = &shared_mem[in_channels * out_channels * kernel_h * kernel_w];
    
    const int tid = threadIdx.x;
    const int lane_id = tid % WARP_SIZE;
    const int warp_id = tid / WARP_SIZE;
    const int warps_per_block = blockDim.x / WARP_SIZE;
    
    // Load weights into shared memory
    for (int i = tid; i < in_channels * out_channels * kernel_h * kernel_w; i += blockDim.x) {
        shared_weight[i] = weight[i];
    }
    __syncthreads();

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total = N * out_channels * H_pool_out * W_pool_out;
    if (idx >= total) return;

    const int w_pool_out = idx % W_pool_out;
    int temp = idx / W_pool_out;
    const int h_pool_out = temp % H_pool_out;
    temp /= H_pool_out;
    const int c_out = temp % out_channels;
    const int n = temp / out_channels;

    float max_val = -FLT_MAX;
    float sum_val = 0.0f;
    int valid_count = 0;

    // Compute convolution and max pooling
    for (int ph = 0; ph < pool_kernel; ph++) {
        for (int pw = 0; pw < pool_kernel; pw++) {
            float conv_val = 0.0f;
            const int h_out = h_pool_out * pool_stride + ph;
            const int w_out = w_pool_out * pool_stride + pw;
            
            for (int c_in = 0; c_in < in_channels; c_in++) {
                for (int kh = 0; kh < kernel_h; kh++) {
                    for (int kw = 0; kw < kernel_w; kw++) {
                        int h_in = (h_out + padding - kh) / stride;
                        int w_in = (w_out + padding - kw) / stride;
                        bool valid = ((h_out + padding - kh) % stride == 0) && 
                                   ((w_out + padding - kw) % stride == 0) &&
                                   (h_in >= 0 && h_in < H_in && w_in >= 0 && w_in < W_in);

                        if (valid) {
                            const int input_idx = ((n * in_channels + c_in) * H_in + h_in) * W_in + w_in;
                            const int weight_idx = ((c_in * out_channels + c_out) * kernel_h + kh) * kernel_w + kw;
                            conv_val += input[input_idx] * shared_weight[weight_idx];
                        }
                    }
                }
            }
            conv_val += bias[c_out];
            max_val = max(max_val, conv_val);
            sum_val += conv_val;
            valid_count++;
        }
    }

    // Warp-level reduction for max pooling
    max_val = warp_reduce_max(max_val);
    if (lane_id == 0) {
        output[idx] = max_val;
    }

    // Compute mean using warp-level reduction
    sum_val = warp_reduce_sum(sum_val);
    if (lane_id == 0) {
        shared_reduce[warp_id] = sum_val / valid_count;
    }
    __syncthreads();

    // Final reduction for mean across warps
    if (warp_id == 0 && lane_id < warps_per_block) {
        float mean_val = shared_reduce[lane_id];
        mean_val = warp_reduce_sum(mean_val) / warps_per_block;
        if (lane_id == 0) {
            mean_output[n * out_channels + c_out] = mean_val;
        }
    }
}

torch::Tensor forward(
    torch::Tensor x,
    int64_t stride,
    int64_t padding,
    int64_t maxpool_kernel_size,
    int64_t maxpool_stride,
    double hardtanh_min,
    double hardtanh_max,
    torch::Tensor conv_transpose,
    torch::Tensor conv_transpose_bias
) {
    const int N = x.size(0);
    const int in_channels = x.size(1);
    const int H_in = x.size(2);
    const int W_in = x.size(3);
    const int out_channels = conv_transpose.size(1);
    const int kernel_h = conv_transpose.size(2);
    const int kernel_w = conv_transpose.size(3);

    const int H_conv = (H_in - 1) * stride - 2 * padding + kernel_h;
    const int W_conv = (W_in - 1) * stride - 2 * padding + kernel_w;
    const int H_pool = (H_conv - maxpool_kernel_size) / maxpool_stride + 1;
    const int W_pool = (W_conv - maxpool_kernel_size) / maxpool_stride + 1;

    auto pool_out = torch::empty({N, out_channels, H_pool, W_pool}, x.options());
    auto mean_out = torch::empty({N, out_channels, 1, 1}, x.options());

    const int threads = 256;
    const int total = N * out_channels * H_pool * W_pool;
    const int blocks = (total + threads - 1) / threads;
    
    const int shared_mem_size = (in_channels * out_channels * kernel_h * kernel_w + 
                                threads / WARP_SIZE) * sizeof(float);

    conv_transpose_maxpool_mean_kernel<<<blocks, threads, shared_mem_size>>>(
        x.data_ptr<float>(),
        conv_transpose.data_ptr<float>(),
        conv_transpose_bias.data_ptr<float>(),
        pool_out.data_ptr<float>(),
        mean_out.data_ptr<float>(),
        N, in_channels, H_in, W_in,
        out_channels, kernel_h, kernel_w,
        stride, padding,
        H_conv, W_conv,
        maxpool_kernel_size, maxpool_stride,
        H_pool, W_pool
    );

    pool_out = torch::clamp(pool_out, hardtanh_min, hardtanh_max);
    mean_out = torch::tanh(mean_out);

    return mean_out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Optimized Reduction Forward");
}
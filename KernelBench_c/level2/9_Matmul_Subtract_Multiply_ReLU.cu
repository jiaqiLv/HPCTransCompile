#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for combined linear, subtract, multiply and ReLU operations
template <typename scalar_t>
__global__ void linear_subtract_multiply_relu_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    const int batch_size,
    const int in_features,
    const int out_features,
    const float subtract_value,
    const float multiply_value) {

    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < batch_size && col < out_features) {
        scalar_t sum = 0;
        
        // Compute linear transformation
        for (int k = 0; k < in_features; k++) {
            sum += input[row * in_features + k] * weight[col * in_features + k];
        }
        
        // Add bias
        sum += bias[col];
        
        // Subtract and multiply
        sum = (sum - subtract_value) * multiply_value;
        
        // ReLU activation
        sum = sum > 0 ? sum : 0;
        
        output[row * out_features + col] = sum;
    }
}

torch::Tensor forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    float subtract_value,
    float multiply_value) {
    
    auto batch_size = input.size(0);
    auto in_features = input.size(1);
    auto out_features = weight.size(0);

    auto output = torch::empty({batch_size, out_features}, input.options());

    const dim3 threads(16, 16);
    const dim3 blocks(
        (batch_size + threads.x - 1) / threads.x,
        (out_features + threads.y - 1) / threads.y
    );

    AT_DISPATCH_FLOATING_TYPES(input.type(), "linear_subtract_multiply_relu_kernel", ([&] {
        linear_subtract_multiply_relu_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            batch_size,
            in_features, 
            out_features,
            subtract_value,
            multiply_value
        );
    }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Linear transform with subtract, multiply and ReLU forward");
}
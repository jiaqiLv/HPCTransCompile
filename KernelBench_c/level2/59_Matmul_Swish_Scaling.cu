#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void swish_scaling_kernel_2d(const float* __restrict__ input, float* output, float scaling_factor, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        float x = input[idx];
        // Swish activation: x * sigmoid(x)
        float sigmoid = 1.0f / (1.0f + expf(-x));
        float y = x * sigmoid * scaling_factor;
        output[idx] = y;
    }
}

torch::Tensor forward(
    torch::Tensor x,
    torch::Tensor weight,
    torch::Tensor bias,
    double scaling_factor) {

    // Ensure tensors are contiguous
    x = x.contiguous();
    weight = weight.contiguous();
    bias = bias.contiguous();

    // Ensure tensors are on CUDA
    TORCH_CHECK(x.is_cuda(), "Input tensor 'x' must be a CUDA tensor.");
    TORCH_CHECK(weight.is_cuda(), "Weight tensor must be a CUDA tensor.");
    TORCH_CHECK(bias.is_cuda(), "Bias tensor must be a CUDA tensor.");

    // Ensure data types are float32
    TORCH_CHECK(x.scalar_type() == at::kFloat, "Input tensor 'x' must be of type torch.float32.");
    TORCH_CHECK(weight.scalar_type() == at::kFloat, "Weight tensor must be of type torch.float32.");
    TORCH_CHECK(bias.scalar_type() == at::kFloat, "Bias tensor must be of type torch.float32.");

    // Compute linear transformation: y = x @ weight.T + bias
    auto y = at::addmm(bias, x, weight.t());

    // Get the dimensions
    int rows = y.size(0);
    int cols = y.size(1);

    // Allocate output tensor
    auto output = at::empty_like(y);

    // Launch the CUDA kernel
    dim3 threads(32, 32);
    dim3 blocks((cols + threads.x - 1) / threads.x, (rows + threads.y - 1) / threads.y);

    swish_scaling_kernel_2d<<<blocks, threads>>>(
        y.data_ptr<float>(),
        output.data_ptr<float>(),
        static_cast<float>(scaling_factor),
        rows,
        cols);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "Custom CUDA forward function");
}
